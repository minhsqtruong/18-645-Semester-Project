#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "ctable.h"
#include "image.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__device__ unsigned int g_counter = 0;


// This function returns 
// 1 if v is greater than x + th
// 2 if v is less than x - th
// 0 if v is between x + th and x - th
__device__ __forceinline__ int diffType(const int v, const int x, const int th)
{
    const int diff = x - v;
    return static_cast<int>(diff < -th) + (static_cast<int>(diff > th) << 1);
}


// Integer division with result round up
__host__ __device__ __forceinline__ int divUp(int total, int grain)
{
    return (total + grain - 1) / grain;
}


// mask1/2 light/dark
__device__ void calcMask(const int C[4], const int v, const int th, int& mask1, int& mask2)
{
    mask1 = 0;  // only cares about bright one
    mask2 = 0;  // only cares about dark

    int d1, d2;



    d1 = diffType(v, C[0] & 0xff, th);
    d2 = diffType(v, C[2] & 0xff, th);

    if ((d1 | d2) == 0) // if both sides are between the thresholds
        return;

    mask1 |= (d1 & 1) << 0;
    
    // because we're shifting 2'b10 left, we need to shift one back right
    mask2 |= ((d1 & 2) >> 1) << 0;

    mask1 |= (d2 & 1) << 8;
    mask2 |= ((d2 & 2) >> 1) << 8;



    d1 = diffType(v, C[1] & 0xff, th);
    d2 = diffType(v, C[3] & 0xff, th);

    if ((d1 | d2) == 0)
        return;

    mask1 |= (d1 & 1) << 4;
    mask2 |= ((d1 & 2) >> 1) << 4;

    mask1 |= (d2 & 1) << 12;
    mask2 |= ((d2 & 2) >> 1) << 12;

    // end of four corners

    d1 = diffType(v, (C[0] >> (2 * 8)) & 0xff, th);
    d2 = diffType(v, (C[2] >> (2 * 8)) & 0xff, th);

    if ((d1 | d2) == 0)
        return;

    mask1 |= (d1 & 1) << 2;
    mask2 |= ((d1 & 2) >> 1) << 2;

    mask1 |= (d2 & 1) << 10;
    mask2 |= ((d2 & 2) >> 1) << 10;



    d1 = diffType(v, (C[1] >> (2 * 8)) & 0xff, th);
    d2 = diffType(v, (C[3] >> (2 * 8)) & 0xff, th);

    if ((d1 | d2) == 0)
        return;

    mask1 |= (d1 & 1) << 6;
    mask2 |= ((d1 & 2) >> 1) << 6;

    mask1 |= (d2 & 1) << 14;
    mask2 |= ((d2 & 2) >> 1) << 14;



    d1 = diffType(v, (C[0] >> (1 * 8)) & 0xff, th);
    d2 = diffType(v, (C[2] >> (1 * 8)) & 0xff, th);

    /*if ((d1 | d2) == 0)
        return;*/

    mask1 |= (d1 & 1) << 1;
    mask2 |= ((d1 & 2) >> 1) << 1;

    mask1 |= (d2 & 1) << 9;
    mask2 |= ((d2 & 2) >> 1) << 9;



    d1 = diffType(v, (C[0] >> (3 * 8)) & 0xff, th);
    d2 = diffType(v, (C[2] >> (3 * 8)) & 0xff, th);

    /*if ((d1 | d2) == 0)
        return;*/

    mask1 |= (d1 & 1) << 3;
    mask2 |= ((d1 & 2) >> 1) << 3;

    mask1 |= (d2 & 1) << 11;
    mask2 |= ((d2 & 2) >> 1) << 11;



    d1 = diffType(v, (C[1] >> (1 * 8)) & 0xff, th);
    d2 = diffType(v, (C[3] >> (1 * 8)) & 0xff, th);

    /*if ((d1 | d2) == 0)
        return;*/

    mask1 |= (d1 & 1) << 5;
    mask2 |= ((d1 & 2) >> 1) << 5;

    mask1 |= (d2 & 1) << 13;
    mask2 |= ((d2 & 2) >> 1) << 13;



    d1 = diffType(v, (C[1] >> (3 * 8)) & 0xff, th);
    d2 = diffType(v, (C[3] >> (3 * 8)) & 0xff, th);

    mask1 |= (d1 & 1) << 7;
    mask2 |= ((d1 & 2) >> 1) << 7;

    mask1 |= (d2 & 1) << 15;
    mask2 |= ((d2 & 2) >> 1) << 15;
}

// 1 -> v > x + th
// 2 -> v < x - th
// 0 -> not a keypoint

// popc counts the number of 1's
__device__ __forceinline__ bool isKeyPoint(int mask1, int mask2)
{
    return (__popc(mask1) > 8 && (c_table[(mask1 >> 3) - 63] & (1 << (mask1 & 7)))) ||
           (__popc(mask2) > 8 && (c_table[(mask2 >> 3) - 63] & (1 << (mask2 & 7))));
}

// This is my kernel
__global__ void calcKeyPoints(uint8_t* image, int rows, int cols, int threshold, float *data)
{
    
    const int j = threadIdx.x + blockIdx.x * blockDim.x + 10;
    const int i = threadIdx.y + blockIdx.y * blockDim.y + 10;
    
    
    
        
    if (i < rows - 10 && j < cols - 10)
    {
        
        
        int v;
        int C[4] = {0,0,0,0};
        C[2] |= static_cast<uint8_t>(image[cols*(i - 3) + (j - 1)]) << 8;
        
        C[2] |= static_cast<uint8_t>(image[cols*(i - 3) + (j)]);
        
        C[1] |= static_cast<uint8_t>(image[cols*(i - 3) + (j + 1)]) << (3 * 8);
        
        C[2] |= static_cast<uint8_t>(image[cols*(i - 2) + (j - 2)]) << (2 * 8);
        C[1] |= static_cast<uint8_t>(image[cols*(i - 2) + (j + 2)]) << (2 * 8);

        C[2] |= static_cast<uint8_t>(image[cols*(i - 1) + (j - 3)]) << (3 * 8);
        C[1] |= static_cast<uint8_t>(image[cols*(i - 1) + (j + 3)]) << 8;

        C[3] |= static_cast<uint8_t>(image[cols * (i) + (j - 3)]);
        v     = static_cast<uint8_t>(image[cols * (i) + (j)]);
        C[1] |= static_cast<uint8_t>(image[cols * (i) + (j + 3)]);
        // Checking both sides
        int d1 = diffType(v, C[1] & 0xff, threshold);
        int d2 = diffType(v, C[3] & 0xff, threshold);
        if ((d1 | d2) == 0)
        {
            return;
        }
        C[3] |= static_cast<uint8_t>(image[cols * (i + 1) + (j - 3)]) << 8;
        C[0] |= static_cast<uint8_t>(image[cols * (i + 1) + (j + 3)]) << (3 * 8);

        C[3] |= static_cast<uint8_t>(image[cols * (i + 2) + (j - 2)]) << (2 * 8);
        C[0] |= static_cast<uint8_t>(image[cols * (i + 2) + (j + 2)]) << (2 * 8);

        C[3] |= static_cast<uint8_t>(image[cols * (i + 3) + (j - 1)]) << (3 * 8);
        C[0] |= static_cast<uint8_t>(image[cols * (i + 3) + (j)]);
        C[0] |= static_cast<uint8_t>(image[cols * (i + 3) + (j + 1)]) << 8;

        int mask1 = 0;
        int mask2 = 0;

        calcMask(C, v, threshold, mask1, mask2);
        
        
        
        if (isKeyPoint(mask1, mask2))
        {
            const unsigned int ind = atomicInc(&g_counter, (unsigned int)(-1));
            
            int k = 0;

            while (k < 100)
            {
                // Getting the patch
                data[(ind*100)+k] = static_cast<float>(image[cols*(i+4-(k/10))+(j+(-4+(k%10)))]);
                k++;
            }
   
        }
            
    }


}


int main()
{
    hipProfilerStart();
    
    
    int arr_size = 300;      // 300 is good for threshold 50
    int threshold = 50;
    
    dim3 block(32, 8);
    
    int height = 878;
    int width = 750;

    dim3 grid;
    grid.x = divUp(height - 6, block.x);
    grid.y = divUp(width - 6, block.y);
    

    // Copying sample image to device
    uint8_t* img_d;
    hipMalloc(&img_d, height*width*sizeof(uint8_t));
    hipMemcpy(img_d, img_sample, height*width*sizeof(uint8_t), hipMemcpyHostToDevice);
    

    
    // Memory allocation for output data
    float *gpu_data;
    hipMallocManaged(&gpu_data, 100 * arr_size * sizeof(float));
    
    ///////////////////////////////////////////////////////
    // Most important line of the file/////////////////////
    // Launch the kernel//////////////////////////////////////////
    calcKeyPoints<<<grid, block>>>(img_d, width, height, threshold, gpu_data);
    //////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////
    
    hipDeviceSynchronize();
    // Putting in float4
    float4 *data_out;
    hipMallocManaged(&data_out, 24 * arr_size * sizeof(float4));
    
    
    int i = 0;
    int j = 0;
    int k = 0;
    
    while (k < arr_size)
    {
        while (j < 24)
        {
            data_out[24*k+j] = make_float4(gpu_data[100*k+i], gpu_data[100*k+i+1], gpu_data[100*k+i+2], gpu_data[100*k+i+3]);
            i = i + 4;
            j++;
        }
        i = 0;
        j = 0;
        k++;
    }
    
    
    
    // Printing float4
    for (i = 0; i < 24 * arr_size; i++)
    {
        printf("%f %f %f %f\n", data_out[i].x, data_out[i].y, data_out[i].z, data_out[i].w);
    }
    
    
    
    // Free device memory

    hipFree(img_d);
    hipFree(img_sample);
    hipFree(gpu_data);
    hipFree(data_out);

    
    hipGetLastError();
    hipProfilerStop();
    hipDeviceSynchronize();

    return 0;
}